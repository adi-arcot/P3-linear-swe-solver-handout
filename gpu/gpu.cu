#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "../common/common.hpp"
#include "../common/solver.hpp"

// Global variables for device memory
double *h, *u, *v;                     // Main fields
double *dh, *du, *dv;                  // Current derivatives
double *dh1, *du1, *dv1;               // Previous derivatives
double *dh2, *du2, *dv2;               // Second previous derivatives
int nx, ny;
double H, g, dx, dy, dt;
int t = 0;

// Combined kernel for all derivatives
__global__ void compute_all_derivatives_kernel(double *h, double *u, double *v,
                                             double *dh, double *du, double *dv,
                                             int nx, int ny, double H, double g, 
                                             double dx, double dy) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // faster moving index for coalescing
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < nx && j < ny) {
        // Compute all derivatives at once
        dh(i, j) = -H * (du_dx(i, j) + dv_dy(i, j));
        du(i, j) = -g * dh_dx(i, j);
        dv(i, j) = -g * dh_dy(i, j);
    }
}

// Combined kernel for all ghost cells
__global__ void compute_all_ghost_cells_kernel(double *h, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handle horizontal ghost cells
    if (idx < ny) {
        h(nx, idx) = h(0, idx);
    }
    
    // Handle vertical ghost cells
    if (idx < nx) {
        h(idx, ny) = h(idx, 0);
    }
}

// Combined kernel for all boundaries
__global__ void compute_all_boundaries_kernel(double *u, double *v, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handle horizontal boundaries
    if (idx < ny) {
        u(0, idx) = u(nx, idx);
    }
    
    // Handle vertical boundaries
    if (idx < nx) {
        v(idx, 0) = v(idx, ny);
    }
}

// Multistep kernel remains mostly the same but with swapped indices
__global__ void multistep_kernel(double *h, double *u, double *v,
                                double *dh, double *du, double *dv,
                                double *dh1, double *du1, double *dv1,
                                double *dh2, double *du2, double *dv2,
                                int nx, int ny, double dt,
                                double a1, double a2, double a3) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < nx && j < ny) {
        // Update height field
        h(i, j) += (a1 * dh(i, j) + 
                    a2 * dh1(i, j) + 
                    a3 * dh2(i, j)) * dt;
        
        // Update u velocity field
        if (i < nx - 1) {
            u(i + 1, j) += (a1 * du(i, j) + 
                           a2 * du1(i, j) + 
                           a3 * du2(i, j)) * dt;
        }
        
        // Update v velocity field
        if (j < ny - 1) {
            v(i, j + 1) += (a1 * dv(i, j) + 
                           a2 * dv1(i, j) + 
                           a3 * dv2(i, j)) * dt;
        }
    }
}

void swap_buffers() {
    double *tmp;
    
    tmp = dh2;
    dh2 = dh1;
    dh1 = dh;
    dh = tmp;
    
    tmp = du2;
    du2 = du1;
    du1 = du;
    du = tmp;
    
    tmp = dv2;
    dv2 = dv1;
    dv1 = dv;
    dv = tmp;
}

void init(double *h0, double *u0, double *v0, double length_, double width_, 
          int nx_, int ny_, double H_, double g_, double dt_, int rank_, int num_procs_) {
    nx = nx_;
    ny = ny_;
    H = H_;
    g = g_;
    dx = length_ / nx;
    dy = width_ / ny;
    dt = dt_;

    // Allocate device memory
    hipMalloc(&h, (nx + 1) * (ny + 1) * sizeof(double));
    hipMalloc(&u, (nx + 1) * ny * sizeof(double));
    hipMalloc(&v, nx * (ny + 1) * sizeof(double));
    
    hipMalloc(&dh, nx * ny * sizeof(double));
    hipMalloc(&du, nx * ny * sizeof(double));
    hipMalloc(&dv, nx * ny * sizeof(double));
    
    hipMalloc(&dh1, nx * ny * sizeof(double));
    hipMalloc(&du1, nx * ny * sizeof(double));
    hipMalloc(&dv1, nx * ny * sizeof(double));
    
    hipMalloc(&dh2, nx * ny * sizeof(double));
    hipMalloc(&du2, nx * ny * sizeof(double));
    hipMalloc(&dv2, nx * ny * sizeof(double));

    // Copy initial conditions to device
    hipMemcpy(h, h0, (nx + 1) * (ny + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(u, u0, (nx + 1) * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(v, v0, nx * (ny + 1) * sizeof(double), hipMemcpyHostToDevice);
    
    // Initialize derivative arrays to zero
    hipMemset(dh, 0, nx * ny * sizeof(double));
    hipMemset(du, 0, nx * ny * sizeof(double));
    hipMemset(dv, 0, nx * ny * sizeof(double));
    hipMemset(dh1, 0, nx * ny * sizeof(double));
    hipMemset(du1, 0, nx * ny * sizeof(double));
    hipMemset(dv1, 0, nx * ny * sizeof(double));
    hipMemset(dh2, 0, nx * ny * sizeof(double));
    hipMemset(du2, 0, nx * ny * sizeof(double));
    hipMemset(dv2, 0, nx * ny * sizeof(double));
}

void step() {
    dim3 blockDim(16, 16);
    dim3 gridDim((ny + blockDim.x - 1) / blockDim.x,  // Swapped nx/ny for coalescing
                 (nx + blockDim.y - 1) / blockDim.y);
    
    // Combined ghost cells kernel
    compute_all_ghost_cells_kernel<<<(max(nx,ny) + 255) / 256, 256>>>(h, nx, ny);
    hipDeviceSynchronize();
    
    // Combined derivatives kernel
    compute_all_derivatives_kernel<<<gridDim, blockDim>>>(h, u, v, dh, du, dv, 
                                                         nx, ny, H, g, dx, dy);
    hipDeviceSynchronize();
    
    // Set multistep coefficients
    double a1, a2 = 0.0, a3 = 0.0;
    if (t == 0) {
        a1 = 1.0;
    } else if (t == 1) {
        a1 = 3.0 / 2.0;
        a2 = -1.0 / 2.0;
    } else {
        a1 = 23.0 / 12.0;
        a2 = -16.0 / 12.0;
        a3 = 5.0 / 12.0;
    }
    
    // Multistep kernel
    multistep_kernel<<<gridDim, blockDim>>>(h, u, v, 
                                           dh, du, dv,
                                           dh1, du1, dv1,
                                           dh2, du2, dv2,
                                           nx, ny, dt, a1, a2, a3);
    hipDeviceSynchronize();
    
    // Combined boundaries kernel
    compute_all_boundaries_kernel<<<(max(nx,ny) + 255) / 256, 256>>>(u, v, nx, ny);
    hipDeviceSynchronize();
    
    swap_buffers();
    t++;
}

void transfer(double *h_host) {
    hipMemcpy(h_host, h, (nx + 1) * (ny + 1) * sizeof(double), hipMemcpyDeviceToHost);
}

void free_memory() {
    hipFree(h);
    hipFree(u);
    hipFree(v);
    hipFree(dh);
    hipFree(du);
    hipFree(dv);
    hipFree(dh1);
    hipFree(du1);
    hipFree(dv1);
    hipFree(dh2);
    hipFree(du2);
    hipFree(dv2);
}